
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <math.h>

using namespace std;

#define row 1000
#define column 1000

#define threadsPB 32

// Funcion para generar numeros randoms en mi matrix:
void randomsInt(double **& matrix)
{
    for(int i=0;i<row;++i){
	    for(int j=0;j<column;++j){
            matrix[i][j]=rand() % 10 + 1;
        }
    }
}


//Funcion para separar memoria  con CUDA
void createMatrixHostCUDA(double**& host, double**& device, double **& aux, int size, int r, int c ){
    host = (double **)malloc(r*sizeof(double*));
    host[0]=(double *)malloc(size);
    aux =(double **)malloc(r*sizeof(double*));

    hipMalloc((void **)&aux[0],size);
    hipMalloc((void **)&device,r*sizeof(double*));

    for (int i=1; i<r;++i){
        host[i]=host[i-1]+c;
        aux[i]=aux[i-1]+c;
    }
    hipMemcpy(device, aux, r*sizeof(double*), hipMemcpyHostToDevice);
}


// Kernel de la funcion sigmoid:
__global__ void sigmoid(double ** A,double ** C) {
     
     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <row && j <column){
         C[i][j]= 1.0/(1+exp(-A[i][j]));
//       printf("i: %i\t j: %i\n" ,i,j );
//       printf("2: %f \n" ,A[i][j] );
     }
}

// Kerndel de la funcion SigmoidGradient:
__global__ void sigmoidGradient(double ** A, double ** C) {
     
     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <row && j <column){
         C[i][j]= (1.0/(1+exp(-A[i][j])))*(1-(1.0/(1+exp(-A[i][j]))));
     }
}

// Kernel de la funcion Multiplicacion por un escalar:
__global__ void scalarMult(double ** A, double s,double ** C) {
     
     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;
     if (i <row && j <column){
         C[i][j]= A[i][j]* s;
//       printf("i: %i\t j: %i\n" ,i,j );
//       printf("2: %f \n" ,A[i][j] );
     }     
}

// Kernel de la funcion Multiplicacion de matrices:
__global__ void Multi(double **A, double **B, double **C){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int suma = 0;

    if(i<row && j<column){
        for(int w=0;w<column;w++){
             suma += A[i][w] * B[w][j];
        }
        C[i][j] = suma;
    }
}

// Funcion imprimir:
void print(double ** a){
	for(int i=0;i<row;++i){
	    for(int j=0;j<column;++j){
            cout<<a[i][j]<<'\t';
        }       
	cout<<endl;
    }

	cout<<endl;
}


int main()
{
	srand (time(NULL));
	double **a, **b, **c;
	double **d_a, **d_b, **d_c;
	double **a_aux, **b_aux, **c_aux;
	int size = row* column * sizeof(double*);

	
	createMatrixHostCUDA(a,d_a,a_aux,size,row,column);
    createMatrixHostCUDA(b,d_b,b_aux,size,row,column);
	createMatrixHostCUDA(c,d_c,c_aux,size,row,column);
	
    randomsInt(a);
    randomsInt(b);

	hipMemcpy(a_aux[0], a[0], size, hipMemcpyHostToDevice);
    hipMemcpy(b_aux[0], b[0], size, hipMemcpyHostToDevice);

	
	dim3 threadPerBlock(threadsPB, threadsPB);
	dim3 blockPerGrid((row+threadPerBlock.x-1)/threadPerBlock.x,(column+threadPerBlock.y-1)/threadPerBlock.y);

    //start_tiempo:
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

 
    hipEventRecord(start,0);
    Multi<<<blockPerGrid,threadPerBlock>>>(d_a,d_b,d_c);
    hipEventRecord(end,0);
    hipEventSynchronize(end);
    float elapsedTime;

    hipEventElapsedTime(&elapsedTime,start,end);
    cout<<"El tiempo es:   "<<elapsedTime<<endl;
    hipEventDestroy(start);
    hipEventDestroy(end);
    //tiempo:





    hipMemcpy(c[0],c_aux[0], size, hipMemcpyDeviceToHost);
	
	hipFree(d_a);hipFree(d_c);
	hipFree(a_aux[0]);hipFree(c_aux[0]);


    cout<<"----A----"<<endl;
	//print (a);

    cout<<"----B----"<<endl;
    //print(b);
    
	cout<<"----c----"<<endl;
	//print(c);

	free(a); free(c);
	return 0;
}